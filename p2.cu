
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define block_size 32

#define n 1024


__global__ void mul_matrix(int *a, int *b, int *c){
    int row = threadIdx.y;
    int col = threadIdx.x;
	int my_x = blockIdx.x*blockDim.x + threadIdx.x;
    int my_y = blockIdx.y*blockDim.y + threadIdx.y;
    __shared__ int A_s[32][32];
    __shared__ int B_s[32][32];
    int local_c = 0;
    int i,j;

    for(i = 0; i < n/block_size ;i++)
    {
        A_s[row][col] = a[my_x*n + (i*blockDim.y + col)];
        B_s[row][col] = b[(i*blockDim.x+row)*n + my_y];
        __syncthreads();
        for(j = 0; j < block_size; j++)
        {
            local_c += A_s[row][j] * B_s[j][col];
        }
        __syncthreads();
    }
    c[my_x*n+my_y] = local_c;
}

int main(){		
    int i;
    int *a = (int*)malloc(sizeof(int)*n*n);
    int *b = (int*)malloc(sizeof(int)*n*n);
    int *c = (int*)malloc(sizeof(int)*n*n);
	
	  for(i=0; i<n*n; i++){
			a[i]=1;
			b[i]=2;
            c[i]=0;
  		}
		int *gpu_a, *gpu_b, *gpu_c;
		hipMalloc((void**)&gpu_a, sizeof(int)*n*n);
		hipMalloc((void**)&gpu_b, sizeof(int)*n*n);
		hipMalloc((void**)&gpu_c, sizeof(int)*n*n);
		
		struct timespec start, stop; 
	    double time;
	  
		hipMemcpy(gpu_a, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
		hipMemcpy(gpu_b, b, sizeof(int)*n*n, hipMemcpyHostToDevice);
		dim3 dimGrid(32,32);
		dim3 dimBlock(32,32);
		
		if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}

		mul_matrix<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
		
		hipMemcpy(c, gpu_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
		
		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
		printf("time is %f ns\n", time*1e9);	 
		
        printf("C[451][451]=%d\n",c[1024*451 + 451]);

		free(a);
		free(b);
		free(c);
		hipFree(gpu_a);  
		hipFree(gpu_b);  
		hipFree(gpu_c);  
		return 0;
}	