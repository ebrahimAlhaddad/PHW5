
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define n 1024


__global__ void matrix_mul(int *a, int *b, int *c){
    int my_x;
    int my_y;
	my_x = blockIdx.x*blockDim.x + threadIdx.x;
    my_y = blockIdx.y*blockDim.y + threadIdx.y;
    int local_c = 0;
    int i;
    for(i = 0; i < n;i++)
    {
        local_c += a[my_x*n+i]*b[i*n+my_y];
    }
    c[my_x*n+my_y] = local_c;
}

int main(){		
    int i;
    int *a = (int*)malloc(sizeof(int)*n*n);
    int *b = (int*)malloc(sizeof(int)*n*n);
    int *c = (int*)malloc(sizeof(int)*n*n);
	
	  for(i=0; i<n*n; i++){
			a[i]=1;
			b[i]=2;
            c[i]=0;
  		}
		int *gpu_a, *gpu_b, *gpu_c;
		hipMalloc((void**)&gpu_a, sizeof(int)*n*n);
		hipMalloc((void**)&gpu_b, sizeof(int)*n*n);
		hipMalloc((void**)&gpu_c, sizeof(int)*n*n);

		struct timespec start, stop; 
	    double time;
		hipMemcpy(gpu_a, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
		hipMemcpy(gpu_b, b, sizeof(int)*n*n, hipMemcpyHostToDevice);
		
		dim3 dimGrid(64,64);
		dim3 dimBlock(16,16);
		
		if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
		
		matrix_mul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
		hipMemcpy(c, gpu_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
		
		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
		printf("Execution time %f ns\n", time*1e9);	 
		
        printf("C[451][451]= %d\n", c[1024*451 + 451]);
  	
		free(a);
		free(b);
		free(c);
		hipFree(gpu_a);  
		hipFree(gpu_b);  
		hipFree(gpu_c);  
		return 0;
}	